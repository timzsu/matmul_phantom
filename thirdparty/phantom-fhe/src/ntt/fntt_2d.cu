#include "hip/hip_runtime.h"
#include "ntt.cuh"
#include "butterfly.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

__global__ static void
inplace_fnwt_radix8_phase1(uint64_t *inout,
                           const uint64_t *twiddles,
                           const uint64_t *twiddles_shoup,
                           const DModulus *modulus,
                           size_t coeff_mod_size,
                           size_t start_mod_idx,
                           size_t n,
                           size_t n1,
                           size_t pad) {
    extern __shared__ uint64_t buffer[];

    // pad address
    size_t pad_tid = threadIdx.x % pad;
    size_t pad_idx = threadIdx.x / pad;

    size_t group = n1 / 8;
    // size of a block
    uint64_t samples[8];
    size_t t = n / 2;

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < n / 8 * coeff_mod_size;
         tid += blockDim.x * gridDim.x) {

        // modulus idx
        size_t twr_idx = tid / (n / 8) + start_mod_idx;
        // index in n/8 range (in each tower)
        size_t n_idx = tid % (n / 8);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + twr_idx * n;
        const uint64_t *psi_shoup = twiddles_shoup + twr_idx * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx].value();
        size_t n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = 1;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + pad_idx + group * j] = samples[j];
        }
        size_t remain_iters = 0;
        __syncthreads();
        for (size_t j = 8, k = group / 2; j < group + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == group / 2)
                remain_iters = 1;
            if (j == group / 4)
                remain_iters = 2;
            __syncthreads();
        }

        if (group < 8)
            remain_iters = (group == 4) ? 2 : 1;
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l];
        }
        if (remain_iters == 1) {
            size_t tw_idx2 = 4 * group * tw_idx + 4 * pad_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (remain_iters == 2) {
            size_t tw_idx2 = 2 * group * tw_idx + 2 * pad_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
        for (size_t l = 0; l < 8; l++) {
            buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l] = samples[l];
        }

        __syncthreads();
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = buffer[pad_tid * (n1 + pad) + pad_idx + group * j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase2(uint64_t *inout,
                           const uint64_t *twiddles,
                           const uint64_t *twiddles_shoup,
                           const DModulus *modulus,
                           size_t coeff_mod_size,
                           size_t start_mod_idx,
                           size_t n,
                           size_t n1,
                           size_t n2) {
    extern __shared__ uint64_t buffer[];

    size_t group = n2 / 8;
    size_t set = threadIdx.x / group;
    // size of a block
    uint64_t samples[8];
    size_t t = n2 / 2;

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (n / 8 * coeff_mod_size);
         tid += blockDim.x * gridDim.x) {

        // prime idx
        size_t twr_idx = coeff_mod_size - 1 - (tid / (n / 8)) + start_mod_idx;
        // index in n/2 range
        size_t n_idx = tid % (n / 8);
        // tid'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx].value();
        const uint64_t *psi = twiddles + n * twr_idx;
        const uint64_t *psi_shoup = twiddles_shoup + n * twr_idx;
        size_t n_init = 2 * m_idx * t + t_idx;
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = n1 + m_idx;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

        for (size_t j = 8, k = t / 8; j < t / 4 + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == t / 8)
                tail = 1;
            if (j == t / 16)
                tail = 2;
            __syncthreads();
        }

        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        if (tail == 1) {
            size_t tw_idx2 = t * tw_idx + 4 * t_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (tail == 2) {
            size_t tw_idx2 = (t / 2) * tw_idx + 2 * t_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        __syncthreads();

        uint64_t modulus2 = modulus << 1;
        // final reduction
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
            csub_q(samples[j], modulus2);
            csub_q(samples[j], modulus);
        }
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase1_include_temp_mod(uint64_t *inout,
                                            const uint64_t *twiddles,
                                            const uint64_t *twiddles_shoup,
                                            const DModulus *modulus,
                                            size_t coeff_mod_size,
                                            size_t start_mod_idx,
                                            size_t total_mod_size,
                                            size_t n,
                                            size_t n1,
                                            size_t pad) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < n / 8 * coeff_mod_size;
         tid += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // modulus idx
        size_t twr_idx = tid / (n / 8) + start_mod_idx;
        size_t twr_idx2 = (twr_idx == coeff_mod_size - 1 ? total_mod_size - 1 : twr_idx);
        // index in n/8 range (in each tower)
        size_t n_idx = tid % (n / 8);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + twr_idx2 * n;
        const uint64_t *psi_shoup = twiddles_shoup + twr_idx2 * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = 1;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + pad_idx + group * j] = samples[j];
        }
        size_t remain_iters = 0;
        __syncthreads();
#pragma unroll
        for (size_t j = 8, k = group / 2; j < group + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == group / 2)
                remain_iters = 1;
            if (j == group / 4)
                remain_iters = 2;
            __syncthreads();
        }

        if (group < 8)
            remain_iters = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l];
        }
        if (remain_iters == 1) {
            size_t tw_idx2 = 4 * group * tw_idx + 4 * pad_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (remain_iters == 2) {
            size_t tw_idx2 = 2 * group * tw_idx + 2 * pad_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l] = samples[l];
        }

        __syncthreads();
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = buffer[pad_tid * (n1 + pad) + pad_idx + group * j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase2_include_temp_mod(uint64_t *inout,
                                            const uint64_t *twiddles,
                                            const uint64_t *twiddles_shoup,
                                            const DModulus *modulus,
                                            size_t coeff_mod_size,
                                            size_t start_mod_idx,
                                            size_t total_mod_size,
                                            size_t n,
                                            size_t n1,
                                            size_t n2) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (n / 8 * coeff_mod_size);
         tid += blockDim.x * gridDim.x) {
        size_t group = n2 / 8;
        size_t set = threadIdx.x / group;
        // size of a block
        uint64_t samples[8];
        size_t t = n2 / 2;
        // prime idx
        size_t twr_idx = coeff_mod_size - 1 - (tid / (n / 8)) + start_mod_idx;
        size_t twr_idx2 = (twr_idx == coeff_mod_size - 1 ? total_mod_size - 1 : twr_idx);
        // index in n/2 range
        size_t n_idx = tid % (n / 8);
        // tid'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + n * twr_idx2;
        const uint64_t *psi_shoup = twiddles_shoup + n * twr_idx2;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = 2 * m_idx * t + t_idx;
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = n1 + m_idx;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = 8, k = t / 8; j < t / 4 + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == t / 8)
                tail = 1;
            if (j == t / 16)
                tail = 2;
            __syncthreads();
        }

#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        if (tail == 1) {
            size_t tw_idx2 = t * tw_idx + 4 * t_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (tail == 2) {
            size_t tw_idx2 = (t / 2) * tw_idx + 2 * t_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        __syncthreads();

        uint64_t modulus2 = modulus << 1;
        // final reduction
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
            csub_q(samples[j], modulus2);
            csub_q(samples[j], modulus);
        }
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase1_include_special_mod(uint64_t *inout,
                                               const uint64_t *twiddles,
                                               const uint64_t *twiddles_shoup,
                                               const DModulus *modulus,
                                               size_t coeff_mod_size,
                                               size_t start_mod_idx,
                                               size_t size_QP,
                                               size_t size_P,
                                               size_t n,
                                               size_t n1,
                                               size_t pad) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < n / 8 * coeff_mod_size;
         tid += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // modulus idx
        size_t twr_idx = tid / (n / 8) + start_mod_idx;
        size_t twr_idx2 = (twr_idx >= start_mod_idx + coeff_mod_size - size_P
                           ? size_QP - (start_mod_idx + coeff_mod_size - twr_idx)
                           : twr_idx);
        // index in n/8 range (in each tower)
        size_t n_idx = tid % (n / 8);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + twr_idx2 * n;
        const uint64_t *psi_shoup = twiddles_shoup + twr_idx2 * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = 1;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + pad_idx + group * j] = samples[j];
        }
        size_t remain_iters = 0;
        __syncthreads();
#pragma unroll
        for (size_t j = 8, k = group / 2; j < group + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == group / 2)
                remain_iters = 1;
            if (j == group / 4)
                remain_iters = 2;
            __syncthreads();
        }

        if (group < 8)
            remain_iters = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l];
        }
        if (remain_iters == 1) {
            size_t tw_idx2 = 4 * group * tw_idx + 4 * pad_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (remain_iters == 2) {
            size_t tw_idx2 = 2 * group * tw_idx + 2 * pad_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l] = samples[l];
        }

        __syncthreads();
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = buffer[pad_tid * (n1 + pad) + pad_idx + group * j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase2_include_special_mod(uint64_t *inout,
                                               const uint64_t *twiddles,
                                               const uint64_t *twiddles_shoup,
                                               const DModulus *modulus,
                                               size_t coeff_mod_size,
                                               size_t start_mod_idx,
                                               size_t size_QP,
                                               size_t size_P,
                                               size_t n,
                                               size_t n1,
                                               size_t n2) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (n / 8 * coeff_mod_size);
         tid += blockDim.x * gridDim.x) {
        size_t group = n2 / 8;
        size_t set = threadIdx.x / group;
        // size of a block
        uint64_t samples[8];
        size_t t = n2 / 2;
        // prime idx
        size_t twr_idx = coeff_mod_size - 1 - (tid / (n / 8)) + start_mod_idx;
        size_t twr_idx2 = (twr_idx >= start_mod_idx + coeff_mod_size - size_P
                           ? size_QP - (start_mod_idx + coeff_mod_size - twr_idx)
                           : twr_idx);
        // index in n/2 range
        size_t n_idx = tid % (n / 8);
        // tid'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + n * twr_idx2;
        const uint64_t *psi_shoup = twiddles_shoup + n * twr_idx2;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = 2 * m_idx * t + t_idx;
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = n1 + m_idx;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = 8, k = t / 8; j < t / 4 + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == t / 8)
                tail = 1;
            if (j == t / 16)
                tail = 2;
            __syncthreads();
        }

#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        if (tail == 1) {
            size_t tw_idx2 = t * tw_idx + 4 * t_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (tail == 2) {
            size_t tw_idx2 = (t / 2) * tw_idx + 2 * t_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        __syncthreads();

        uint64_t modulus2 = modulus << 1;
        // final reduction
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
            csub_q(samples[j], modulus2);
            csub_q(samples[j], modulus);
        }
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

void nwt_2d_radix8_forward_inplace(uint64_t *inout,
                                   const DNTTTable &ntt_tables,
                                   size_t coeff_modulus_size,
                                   size_t start_modulus_idx,
                                   const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase1_sample_size = SAMPLE_SIZE(poly_degree);
    const size_t phase2_sample_size = poly_degree / phase1_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    inplace_fnwt_radix8_phase1<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            per_block_pad);
    // max 512 threads per block
    inplace_fnwt_radix8_phase2<<<
    gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size);
}

void nwt_2d_radix8_forward_inplace_include_temp_mod(uint64_t *inout,
                                                    const DNTTTable &ntt_tables,
                                                    size_t coeff_modulus_size,
                                                    size_t start_modulus_idx,
                                                    size_t total_modulus_size,
                                                    const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase1_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase2_sample_size = poly_degree / phase1_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    //
    inplace_fnwt_radix8_phase1_include_temp_mod<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            total_modulus_size,
            poly_degree,
            phase1_sample_size,
            per_block_pad);
    // max 512 threads per block
    inplace_fnwt_radix8_phase2_include_temp_mod<<<
    gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            total_modulus_size,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size);
}

void nwt_2d_radix8_forward_inplace_include_special_mod(uint64_t *inout,
                                                       const DNTTTable &ntt_tables,
                                                       size_t coeff_modulus_size,
                                                       size_t start_modulus_idx,
                                                       size_t size_QP,
                                                       size_t size_P,
                                                       const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase1_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase2_sample_size = poly_degree / phase1_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    //
    inplace_fnwt_radix8_phase1_include_special_mod<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            size_QP,
            size_P,
            poly_degree,
            phase1_sample_size,
            per_block_pad);
    // max 512 threads per block
    inplace_fnwt_radix8_phase2_include_special_mod<<<
    gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            size_QP,
            size_P,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size);
}
