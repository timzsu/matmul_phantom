#include "hip/hip_runtime.h"
#include "ntt.cuh"
#include "butterfly.cuh"
#include "common.h"

using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

// use in key switching mod up
__global__ static void
inplace_fnwt_radix8_phase1_single_mod_mod_up_fuse(uint64_t *out,
                                                  const uint64_t *in,
                                                  const uint64_t *twiddles,
                                                  const uint64_t *twiddles_shoup,
                                                  const DModulus *modulus,
                                                  size_t coeff_mod_size,
                                                  size_t start_mod_idx,
                                                  size_t n,
                                                  size_t n1,
                                                  size_t pad,
                                                  size_t mod_idx) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < n / 8 * coeff_mod_size;
         tid += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // modulus idx
        size_t twr_idx = tid / (n / 8) + start_mod_idx;
        // index in n/8 range (in each tower)
        size_t n_idx = tid % (n / 8);
        const uint64_t *psi = twiddles + mod_idx * n;
        const uint64_t *psi_shoup = twiddles_shoup + mod_idx * n;
        uint64_t mod_value = modulus[mod_idx].value();
        uint64_t barrett_mu_hi = modulus[twr_idx].const_ratio()[1];
        size_t n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

        // base address
        size_t indata_offset = twr_idx * n;
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = in[indata_offset + n_init + t / 4 * j];
            //            samples[j] = barrett_reduce_uint64_uint64(samples[j], mod_value, barrett_mu_hi);
        }

        size_t tw_idx = 1;

        fntt8(samples, psi, psi_shoup, tw_idx, mod_value);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + pad_idx + group * j] = samples[j];
        }
        size_t remain_iters = 0;
        __syncthreads();
#pragma unroll
        for (size_t j = 8, k = group / 2; j < group + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, mod_value);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == group / 2)
                remain_iters = 1;
            if (j == group / 4)
                remain_iters = 2;

            __syncthreads();
        }

        if (group < 8)
            remain_iters = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l];
        }
        if (remain_iters == 1) {
            size_t tw_idx2 = 4 * group * tw_idx + 4 * pad_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], mod_value);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], mod_value);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], mod_value);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], mod_value);
        } else if (remain_iters == 2) {
            size_t tw_idx2 = 2 * group * tw_idx + 2 * pad_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, mod_value);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, mod_value);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l] = samples[l];
        }

        __syncthreads();
        // base address
        uint64_t *data_ptr = out + twr_idx * n;
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = buffer[pad_tid * (n1 + pad) + pad_idx + group * j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase2_single_mod(uint64_t *inout,
                                      const uint64_t *twiddles,
                                      const uint64_t *twiddles_shoup,
                                      const DModulus *modulus,
                                      size_t coeff_mod_size,
                                      size_t start_mod_idx,
                                      size_t n,
                                      size_t n1,
                                      size_t n2,
                                      size_t mod_idx) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (n / 8 * coeff_mod_size);
         tid += blockDim.x * gridDim.x) {
        size_t group = n2 / 8;
        size_t set = threadIdx.x / group;
        // size of a block
        uint64_t samples[8];
        size_t t = n2 / 2;
        // prime idx
        size_t twr_idx = coeff_mod_size - 1 - (tid / (n / 8)) + start_mod_idx;
        // index in n/2 range
        size_t n_idx = tid % (n / 8);
        // tid'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[mod_idx].value();
        const uint64_t *psi = twiddles + n * mod_idx;
        const uint64_t *psi_shoup = twiddles_shoup + n * mod_idx;
        size_t n_init = 2 * m_idx * t + t_idx;
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = n1 + m_idx;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = samples[j];
        }
        size_t tail = 0;

        __syncthreads();

#pragma unroll
        for (size_t j = 8, k = t / 8; j < t / 4 + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == t / 8)
                tail = 1;
            if (j == t / 16)
                tail = 2;

            __syncthreads();
        }

#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        if (tail == 1) {
            size_t tw_idx2 = t * tw_idx + 4 * t_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (tail == 2) {
            size_t tw_idx2 = (t / 2) * tw_idx + 2 * t_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        __syncthreads();

        uint64_t modulus2 = modulus << 1;
        // final reduction
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
            csub_q(samples[j], modulus2);
            csub_q(samples[j], modulus);
        }
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

// fuse in key switching mod up
void nwt_2d_radix8_forward_modup_fuse(uint64_t *out,
                                      const uint64_t *in,
                                      size_t modulus_index,
                                      const DNTTTable &ntt_tables,
                                      size_t coeff_modulus_size,
                                      size_t start_modulus_idx,
                                      const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase1_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase2_sample_size = poly_degree / phase1_sample_size;
    const size_t per_block_memory = phantom::util::blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);

    inplace_fnwt_radix8_phase1_single_mod_mod_up_fuse<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            out,
            in,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            per_block_pad,
            modulus_index);
    // max 512 threads per block
    inplace_fnwt_radix8_phase2_single_mod<<<
    gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            out,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size,
            modulus_index);
}
