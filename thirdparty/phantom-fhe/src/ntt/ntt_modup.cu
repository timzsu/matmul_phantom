#include "hip/hip_runtime.h"
#include "ntt.cuh"
#include "butterfly.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

__global__ static void
inwt_radix8_phase1(uint64_t *out,
                   const uint64_t *in,
                   const uint64_t *itwiddles,
                   const uint64_t *itwiddles_shoup,
                   const DModulus *modulus,
                   const size_t coeff_mod_size,
                   const size_t start_mod_idx,
                   const size_t n,
                   const size_t n1,
                   const size_t n2) {
    extern __shared__ uint64_t buffer[];
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
         i < (n / 8 * coeff_mod_size);
         i += blockDim.x * gridDim.x) {
        size_t group = n2 / 8;
        size_t set = threadIdx.x / group;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2 / n1;
        // prime idx
        size_t twr_idx = i / (n / 8) + start_mod_idx;
        // index in N/2 range
        size_t n_idx = i % (n / 8);
        // i'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        auto in_ptr = in + twr_idx * n;
        auto out_ptr = out + twr_idx * n;
        const uint64_t *psi = itwiddles + n * twr_idx;
        const uint64_t *psi_shoup = itwiddles_shoup + n * twr_idx;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx].value();
        size_t n_init = 2 * m_idx * t + t_idx;

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = *(in_ptr + n_init + t / 4 * j);
        }
        __syncthreads();

#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        size_t tw_idx = n1 + m_idx;
        size_t tw_idx2 = (t / 4) * tw_idx + t_idx;
        intt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = t / 32, k = 32; j > 0; j >>= 3, k *= 8) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            tw_idx2 = j * tw_idx + m_idx2;
            intt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == 2)
                tail = 1;
            if (j == 4)
                tail = 2;
            __syncthreads();
        }

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
        }
        if (tail == 1) {
            gs_butterfly(samples[0], samples[4], psi[tw_idx], psi_shoup[tw_idx], modulus);
            gs_butterfly(samples[1], samples[5], psi[tw_idx], psi_shoup[tw_idx], modulus);
            gs_butterfly(samples[2], samples[6], psi[tw_idx], psi_shoup[tw_idx], modulus);
            gs_butterfly(samples[3], samples[7], psi[tw_idx], psi_shoup[tw_idx], modulus);
        } else if (tail == 2) {
            intt4(samples, psi, psi_shoup, tw_idx, modulus);
            intt4(samples + 1, psi, psi_shoup, tw_idx, modulus);
        }
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(out_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

__global__ static void
inplace_inwt_radix8_phase2(uint64_t *inout,
                           const uint64_t *itwiddles,
                           const uint64_t *itwiddles_shoup,
                           const uint64_t *inv_degree_modulo,
                           const uint64_t *inv_degree_modulo_shoup,
                           const DModulus *modulus,
                           const size_t coeff_mod_size,
                           const size_t start_mod_idx,
                           const size_t n,
                           const size_t n1,
                           const size_t pad) {
    extern __shared__ uint64_t buffer[];
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n / 8 * coeff_mod_size);
         i += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // prime idx
        size_t twr_idx = i / (n / 8) + start_mod_idx;
        // index in N/2 range
        size_t n_idx = i % (n / 8);

        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = itwiddles + n * twr_idx;
        const uint64_t *psi_shoup = itwiddles_shoup + n * twr_idx;
        const DModulus *modulus_table = modulus;
        uint64_t modulus_value = modulus_table[twr_idx].value();
        uint64_t inv_degree_mod = inv_degree_modulo[twr_idx];
        uint64_t inv_degree_mod_shoup = inv_degree_modulo_shoup[twr_idx];

        size_t n_init = 2 * t / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 / group * j);
        }
        size_t tw_idx = 1;
        size_t tw_idx2 = group * tw_idx + pad_idx;
        intt8(samples, psi, psi_shoup, tw_idx2, modulus_value);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + 8 * pad_idx + j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = group / 8, k = 32; j > 0; j >>= 3, k *= 8) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 +
                                    (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            intt8(samples, psi, psi_shoup, tw_idx2, modulus_value);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == 2)
                tail = 1;
            if (j == 4)
                tail = 2;
            __syncthreads();
        }
        if (group < 8)
            tail = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[pad_tid * (n1 + pad) + pad_idx + group * l];
        }
        if (tail == 1) {
            gs_butterfly(samples[0], samples[4], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[1], samples[5], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[2], samples[6], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[3], samples[7], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
        } else if (tail == 2) {
            intt4(samples, psi, psi_shoup, tw_idx, modulus_value);
            intt4(samples + 1, psi, psi_shoup, tw_idx, modulus_value);
        }

        for (size_t j = 0; j < 4; j++) {
            samples[j] =
                    multiply_and_reduce_shoup_lazy(samples[j], inv_degree_mod, inv_degree_mod_shoup, modulus_value);
        }

        n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            csub_q(samples[j], modulus_value);
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

__global__ static void
inplace_inwt_radix8_phase2_scale(uint64_t *inout,
                                 const uint64_t *itwiddles,
                                 const uint64_t *itwiddles_shoup,
                                 const uint64_t *inv_degree_modulo,
                                 const uint64_t *inv_degree_modulo_shoup,
                                 const DModulus *modulus,
                                 const size_t coeff_mod_size,
                                 const size_t start_mod_idx,
                                 const size_t n,
                                 const size_t n1,
                                 const size_t pad,
                                 const uint64_t *scale,
                                 const uint64_t *scale_shoup) {
    extern __shared__ uint64_t buffer[];
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (n / 8 * coeff_mod_size);
         i += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // prime idx
        size_t twr_idx = i / (n / 8) + start_mod_idx;
        // index in N/2 range
        size_t n_idx = i % (n / 8);

        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = itwiddles + n * twr_idx;
        const uint64_t *psi_shoup = itwiddles_shoup + n * twr_idx;
        const DModulus *modulus_table = modulus;
        uint64_t modulus_value = modulus_table[twr_idx].value();
        uint64_t inv_degree_mod = inv_degree_modulo[twr_idx];
        uint64_t inv_degree_mod_shoup = inv_degree_modulo_shoup[twr_idx];

        size_t n_init = 2 * t / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 / group * j);
        }
        size_t tw_idx = 1;
        size_t tw_idx2 = group * tw_idx + pad_idx;
        intt8(samples, psi, psi_shoup, tw_idx2, modulus_value);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + 8 * pad_idx + j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = group / 8, k = 32; j > 0; j >>= 3, k *= 8) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 +
                                    (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            intt8(samples, psi, psi_shoup, tw_idx2, modulus_value);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == 2)
                tail = 1;
            if (j == 4)
                tail = 2;
            __syncthreads();
        }
        if (group < 8)
            tail = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[pad_tid * (n1 + pad) + pad_idx + group * l];
        }
        if (tail == 1) {
            gs_butterfly(samples[0], samples[4], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[1], samples[5], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[2], samples[6], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
            gs_butterfly(samples[3], samples[7], psi[tw_idx], psi_shoup[tw_idx], modulus_value);
        } else if (tail == 2) {
            intt4(samples, psi, psi_shoup, tw_idx, modulus_value);
            intt4(samples + 1, psi, psi_shoup, tw_idx, modulus_value);
        }

        for (size_t j = 0; j < 4; j++) {
            samples[j] =
                    multiply_and_reduce_shoup_lazy(samples[j], inv_degree_mod, inv_degree_mod_shoup, modulus_value);
        }

        n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));
        uint64_t scale_q = scale[twr_idx];
        uint64_t scale_q_shoup = scale_shoup[twr_idx];
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = multiply_and_reduce_shoup(
                    samples[j], scale_q, scale_q_shoup, modulus_value);
        }
    }
}

void nwt_2d_radix8_backward(uint64_t *out,
                            const uint64_t *in,
                            const DNTTTable &ntt_tables,
                            size_t coeff_modulus_size,
                            size_t start_modulus_idx,
                            const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase2_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase1_sample_size = poly_degree / phase2_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    inwt_radix8_phase1<<<gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            out,
            in,
            ntt_tables.itwiddle(),
            ntt_tables.itwiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size);
    inplace_inwt_radix8_phase2<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            out,
            ntt_tables.itwiddle(), ntt_tables.itwiddle_shoup(),
            ntt_tables.n_inv_mod_q(), ntt_tables.n_inv_mod_q_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            per_block_pad);
}

void nwt_2d_radix8_backward_scale(uint64_t *out,
                                  const uint64_t *in,
                                  const DNTTTable &ntt_tables,
                                  size_t coeff_modulus_size,
                                  size_t start_modulus_idx,
                                  const uint64_t *scale,
                                  const uint64_t *scale_shoup,
                                  const hipStream_t &stream) {
    size_t poly_degree = ntt_tables.n();
    size_t phase2_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase1_sample_size = poly_degree / phase2_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    inwt_radix8_phase1<<<gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            out,
            in,
            ntt_tables.itwiddle(),
            ntt_tables.itwiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size);
    inplace_inwt_radix8_phase2_scale<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            out,
            ntt_tables.itwiddle(), ntt_tables.itwiddle_shoup(),
            ntt_tables.n_inv_mod_q(), ntt_tables.n_inv_mod_q_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            poly_degree,
            phase1_sample_size,
            per_block_pad,
            scale, scale_shoup);
}

__global__ static void
inplace_fnwt_radix8_phase1_include_special_mod_exclude_range(uint64_t *inout,
                                                             const uint64_t *twiddles,
                                                             const uint64_t *twiddles_shoup,
                                                             const DModulus *modulus,
                                                             size_t coeff_mod_size,
                                                             size_t start_mod_idx,
                                                             size_t size_QP, size_t size_P,
                                                             size_t n,
                                                             size_t n1,
                                                             size_t pad,
                                                             size_t excluded_range_start, size_t excluded_range_end) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < n / 8 * coeff_mod_size;
         tid += blockDim.x * gridDim.x) {
        // pad address
        size_t pad_tid = threadIdx.x % pad;
        size_t pad_idx = threadIdx.x / pad;

        size_t group = n1 / 8;
        // size of a block
        uint64_t samples[8];
        size_t t = n / 2;
        // modulus idx
        size_t twr_idx = tid / (n / 8) + start_mod_idx;
        if (twr_idx >= excluded_range_start && twr_idx < excluded_range_end) continue;
        size_t twr_idx2 = (twr_idx >= start_mod_idx + coeff_mod_size - size_P
                           ? size_QP - (start_mod_idx + coeff_mod_size - twr_idx)
                           : twr_idx);
        // index in n/8 range (in each tower)
        size_t n_idx = tid % (n / 8);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + twr_idx2 * n;
        const uint64_t *psi_shoup = twiddles_shoup + twr_idx2 * n;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = t / 4 / group * pad_idx + pad_tid + pad * (n_idx / (group * pad));

#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = 1;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[pad_tid * (n1 + pad) + pad_idx + group * j] = samples[j];
        }
        size_t remain_iters = 0;
        __syncthreads();
#pragma unroll
        for (size_t j = 8, k = group / 2; j < group + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = pad_idx / (k / 4);
            size_t t_idx2 = pad_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] = buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[(n1 + pad) * pad_tid + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] = samples[l];
            }
            if (j == group / 2)
                remain_iters = 1;
            if (j == group / 4)
                remain_iters = 2;
            __syncthreads();
        }

        if (group < 8)
            remain_iters = (group == 4) ? 2 : 1;
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l];
        }
        if (remain_iters == 1) {
            size_t tw_idx2 = 4 * group * tw_idx + 4 * pad_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (remain_iters == 2) {
            size_t tw_idx2 = 2 * group * tw_idx + 2 * pad_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[(n1 + pad) * pad_tid + 8 * pad_idx + l] = samples[l];
        }

        __syncthreads();
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = buffer[pad_tid * (n1 + pad) + pad_idx + group * j];
        }
    }
}

__global__ static void
inplace_fnwt_radix8_phase2_include_special_mod_exclude_range(uint64_t *inout,
                                                             const uint64_t *twiddles,
                                                             const uint64_t *twiddles_shoup,
                                                             const DModulus *modulus,
                                                             size_t coeff_mod_size,
                                                             size_t start_mod_idx,
                                                             size_t size_QP, size_t size_P,
                                                             size_t n,
                                                             size_t n1,
                                                             size_t n2,
                                                             size_t excluded_range_start, size_t excluded_range_end) {
    extern __shared__ uint64_t buffer[];

    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (n / 8 * coeff_mod_size);
         tid += blockDim.x * gridDim.x) {
        size_t group = n2 / 8;
        size_t set = threadIdx.x / group;
        // size of a block
        uint64_t samples[8];
        size_t t = n2 / 2;
        // prime idx
        size_t twr_idx = coeff_mod_size - 1 - (tid / (n / 8)) + start_mod_idx;
        if (twr_idx >= excluded_range_start && twr_idx < excluded_range_end) continue;
        size_t twr_idx2 = (twr_idx >= start_mod_idx + coeff_mod_size - size_P
                           ? size_QP - (start_mod_idx + coeff_mod_size - twr_idx)
                           : twr_idx);
        // index in n/2 range
        size_t n_idx = tid % (n / 8);
        // tid'th block
        size_t m_idx = n_idx / (t / 4);
        size_t t_idx = n_idx % (t / 4);
        // base address
        uint64_t *data_ptr = inout + twr_idx * n;
        const uint64_t *psi = twiddles + n * twr_idx2;
        const uint64_t *psi_shoup = twiddles_shoup + n * twr_idx2;
        const DModulus *modulus_table = modulus;
        uint64_t modulus = modulus_table[twr_idx2].value();
        size_t n_init = 2 * m_idx * t + t_idx;
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = *(data_ptr + n_init + t / 4 * j);
        }
        size_t tw_idx = n1 + m_idx;
        fntt8(samples, psi, psi_shoup, tw_idx, modulus);
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            buffer[set * n2 + t_idx + t / 4 * j] = samples[j];
        }
        size_t tail = 0;
        __syncthreads();

#pragma unroll
        for (size_t j = 8, k = t / 8; j < t / 4 + 1; j *= 8, k >>= 3) {
            size_t m_idx2 = t_idx / (k / 4);
            size_t t_idx2 = t_idx % (k / 4);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                samples[l] =
                        buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l];
            }
            size_t tw_idx2 = j * tw_idx + m_idx2;
            fntt8(samples, psi, psi_shoup, tw_idx2, modulus);
#pragma unroll
            for (size_t l = 0; l < 8; l++) {
                buffer[set * n2 + 2 * m_idx2 * k + t_idx2 + (k / 4) * l] =
                        samples[l];
            }
            if (j == t / 8)
                tail = 1;
            if (j == t / 16)
                tail = 2;
            __syncthreads();
        }

#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            samples[l] = buffer[set * n2 + 8 * t_idx + l];
        }
        if (tail == 1) {
            size_t tw_idx2 = t * tw_idx + 4 * t_idx;
            ct_butterfly(samples[0], samples[1], psi[tw_idx2], psi_shoup[tw_idx2], modulus);
            ct_butterfly(samples[2], samples[3], psi[tw_idx2 + 1], psi_shoup[tw_idx2 + 1], modulus);
            ct_butterfly(samples[4], samples[5], psi[tw_idx2 + 2], psi_shoup[tw_idx2 + 2], modulus);
            ct_butterfly(samples[6], samples[7], psi[tw_idx2 + 3], psi_shoup[tw_idx2 + 3], modulus);
        } else if (tail == 2) {
            size_t tw_idx2 = (t / 2) * tw_idx + 2 * t_idx;
            fntt4(samples, psi, psi_shoup, tw_idx2, modulus);
            fntt4(samples + 4, psi, psi_shoup, tw_idx2 + 1, modulus);
        }
#pragma unroll
        for (size_t l = 0; l < 8; l++) {
            buffer[set * n2 + 8 * t_idx + l] = samples[l];
        }
        __syncthreads();

        uint64_t modulus2 = modulus << 1;
        // final reduction
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            samples[j] = buffer[set * n2 + t_idx + t / 4 * j];
            csub_q(samples[j], modulus2);
            csub_q(samples[j], modulus);
        }
#pragma unroll
        for (size_t j = 0; j < 8; j++) {
            *(data_ptr + n_init + t / 4 * j) = samples[j];
        }
    }
}

void nwt_2d_radix8_forward_inplace_include_special_mod_exclude_range(
        uint64_t *inout,
        const DNTTTable &ntt_tables,
        size_t coeff_modulus_size,
        size_t start_modulus_idx,
        size_t size_QP, size_t size_P,
        size_t excluded_range_start, size_t excluded_range_end,
        const hipStream_t &stream) {
    if (excluded_range_start < start_modulus_idx ||
        excluded_range_end > (start_modulus_idx + coeff_modulus_size)) {
        throw std::invalid_argument("Excluded range in NTT is invalid.");
    }

    size_t poly_degree = ntt_tables.n();
    size_t phase1_sample_size = SAMPLE_SIZE(poly_degree);

    const size_t phase2_sample_size = poly_degree / phase1_sample_size;
    const size_t per_block_memory = blockDimNTT.x * per_thread_sample_size * sizeof(uint64_t);
    //
    inplace_fnwt_radix8_phase1_include_special_mod_exclude_range<<<
    gridDimNTT, (phase1_sample_size / 8) * per_block_pad,
    (phase1_sample_size + per_block_pad + 1) * per_block_pad * sizeof(uint64_t), stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            size_QP, size_P,
            poly_degree,
            phase1_sample_size,
            per_block_pad,
            excluded_range_start, excluded_range_end);
    // max 512 threads per block
    inplace_fnwt_radix8_phase2_include_special_mod_exclude_range<<<
    gridDimNTT, blockDimNTT, per_block_memory, stream>>>(
            inout,
            ntt_tables.twiddle(),
            ntt_tables.twiddle_shoup(),
            ntt_tables.modulus(),
            coeff_modulus_size,
            start_modulus_idx,
            size_QP, size_P,
            poly_degree,
            phase1_sample_size,
            phase2_sample_size,
            excluded_range_start, excluded_range_end);
}
