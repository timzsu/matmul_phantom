#include "hip/hip_runtime.h"
#include "util/modulus.h"
#include "util/common.h"
#include "util/uintarith.h"
#include "util/numth.h"
#include "util/uintarithsmallmod.h"
#include "util/globals.h"
#include <numeric>
#include <stdexcept>
#include <unordered_map>

using namespace std;

namespace phantom::arith {
    void Modulus::set_value(uint64_t value) {
        if (value == 0) {
            // Zero settings
            bit_count_ = 0;
            uint64_count_ = 1;
            value_ = 0;
            const_ratio_ = {{0, 0, 0}};
            is_prime_ = false;
        } else if ((value >> MOD_BIT_COUNT_MAX != 0) || (value == 1)) {
            throw invalid_argument("value can be at most 61-bit and cannot be 1");
        } else {
            value_ = value;
            bit_count_ = get_significant_bit_count(value_);

            // Compute Barrett ratios for 64-bit words (barrett_reduce_128)
            uint64_t numerator[3]{0, 0, 1};
            uint64_t quotient[3]{0, 0, 0};

            // quotient = numerator（1<<128）/ value_,
            // numerator = numerator - quotient * value
            divide_uint192_inplace(numerator, value_, quotient);

            const_ratio_[0] = quotient[0];
            const_ratio_[1] = quotient[1];

            // We store also the remainder
            const_ratio_[2] = numerator[0];

            uint64_count_ = 1;

            // Set the primality flag
            is_prime_ = ::phantom::arith::is_prime(*this);
        }
    }

    uint64_t Modulus::reduce(uint64_t value) const {
        if (value_ == 0) {
            throw logic_error("cannot reduce modulo a zero modulus");
        }
        return barrett_reduce_64(value, *this);
    }

    vector <Modulus> CoeffModulus::BFVDefault(size_t poly_modulus_degree, sec_level_type sec_level) {
        if (!MaxBitCount(poly_modulus_degree, sec_level)) {
            throw invalid_argument("non-standard poly_modulus_degree");
        }
        if (sec_level == sec_level_type::none) {
            throw invalid_argument("invalid security level");
        }

        switch (sec_level) {
            case sec_level_type::tc128:
                return util::global_variables::GetDefaultCoeffModulus128().at(poly_modulus_degree);

            case sec_level_type::tc192:
                return util::global_variables::GetDefaultCoeffModulus192().at(poly_modulus_degree);

            case sec_level_type::tc256:
                return util::global_variables::GetDefaultCoeffModulus256().at(poly_modulus_degree);

            default:
                throw runtime_error("invalid security level");
        }
    }

    vector <Modulus> CoeffModulus::Create(size_t poly_modulus_degree, const vector<int> &bit_sizes) {
        if (poly_modulus_degree > POLY_MOD_DEGREE_MAX || poly_modulus_degree < POLY_MOD_DEGREE_MIN ||
            get_power_of_two(static_cast<uint64_t>(poly_modulus_degree)) < 0) {
            throw invalid_argument("poly_modulus_degree is invalid");
        }
        if (bit_sizes.size() > COEFF_MOD_COUNT_MAX) {
            throw invalid_argument("bit_sizes is invalid");
        }
        if (accumulate(
                bit_sizes.cbegin(), bit_sizes.cend(), USER_MOD_BIT_COUNT_MIN,
                [](int a, int b) { return max(a, b); }) > USER_MOD_BIT_COUNT_MAX ||
            accumulate(bit_sizes.cbegin(), bit_sizes.cend(), USER_MOD_BIT_COUNT_MAX,
                       [](int a, int b) { return min(a, b); }) < USER_MOD_BIT_COUNT_MIN) {
            throw invalid_argument("bit_sizes is invalid");
        }

        unordered_map<int, size_t> count_table;
        unordered_map<int, vector<Modulus>> prime_table;
        for (int size: bit_sizes) {
            ++count_table[size];
        }
        for (const auto &table_elt: count_table) {
            prime_table[table_elt.first] = get_primes(poly_modulus_degree, table_elt.first, table_elt.second);
        }

        vector<Modulus> result;
        for (int size: bit_sizes) {
            result.emplace_back(prime_table[size].back());
            prime_table[size].pop_back();
        }
        return result;
    }

    void Modulus::save_members(std::ostream &stream) const {
        auto old_except_mask = stream.exceptions();
        try {
            // Throw exceptions on std::ios_base::badbit and std::ios_base::failbit
            stream.exceptions(ios_base::badbit | ios_base::failbit);

            stream.write(reinterpret_cast<const char *>(&value_), sizeof(uint64_t));
        }
        catch (const ios_base::failure &) {
            stream.exceptions(old_except_mask);
            throw runtime_error("I/O error");
        }
        catch (...) {
            stream.exceptions(old_except_mask);
            throw;
        }
        stream.exceptions(old_except_mask);
    }

    void Modulus::load_members(istream &stream) {
        auto old_except_mask = stream.exceptions();
        try {
            // Throw exceptions on std::ios_base::badbit and std::ios_base::failbit
            stream.exceptions(ios_base::badbit | ios_base::failbit);

            uint64_t value;
            stream.read(reinterpret_cast<char *>(&value), sizeof(uint64_t));
            set_value(value);
        }
        catch (const ios_base::failure &) {
            stream.exceptions(old_except_mask);
            throw runtime_error("I/O error");
        }
        catch (...) {
            stream.exceptions(old_except_mask);
            throw;
        }
        stream.exceptions(old_except_mask);
    }
}
