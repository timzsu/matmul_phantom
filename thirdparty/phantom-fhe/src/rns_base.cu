#include "hip/hip_runtime.h"
#include "ntt.cuh"
#include "polymath.cuh"
#include "rns.cuh"
#include "rns_base.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

namespace phantom::arith {

    void DRNSBase::init(const RNSBase &cpu_rns_base, const hipStream_t &stream) {
        size_ = cpu_rns_base.size();

        base_ = phantom::util::make_cuda_auto_ptr<DModulus>(size_, stream);
        for (size_t idx = 0; idx < size_; idx++) {
            auto temp_modulus = *(cpu_rns_base.base() + idx);
            DModulus temp(temp_modulus.value(), temp_modulus.const_ratio().at(0), temp_modulus.const_ratio().at(1));
            hipMemcpyAsync(base() + idx, &temp, sizeof(DModulus),
                            hipMemcpyHostToDevice, stream);
        }

        big_Q_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_, stream);
        hipMemcpyAsync(big_modulus(), cpu_rns_base.big_modulus(), size_ * sizeof(uint64_t),
                        hipMemcpyHostToDevice, stream);

        big_qiHat_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_ * size_, stream);
        hipMemcpyAsync(big_qiHat(), cpu_rns_base.big_qiHat(), size_ * size_ * sizeof(std::uint64_t),
                        hipMemcpyHostToDevice, stream);

        qiHat_mod_qi_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_, stream);
        qiHat_mod_qi_shoup_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_, stream);
        hipMemcpyAsync(qiHat_mod_qi_.get(), cpu_rns_base.qiHat_mod_qi(), size_ * sizeof(uint64_t),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(qiHat_mod_qi_shoup_.get(), cpu_rns_base.qiHat_mod_qi_shoup(), size_ * sizeof(uint64_t),
                        hipMemcpyHostToDevice, stream);

        qiHatInv_mod_qi_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_, stream);
        qiHatInv_mod_qi_shoup_ = phantom::util::make_cuda_auto_ptr<uint64_t>(size_, stream);
        hipMemcpyAsync(qiHatInv_mod_qi_.get(), cpu_rns_base.QHatInvModq(), size_ * sizeof(uint64_t),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(qiHatInv_mod_qi_shoup_.get(), cpu_rns_base.QHatInvModq_shoup(), size_ * sizeof(uint64_t),
                        hipMemcpyHostToDevice, stream);

        qiInv_ = phantom::util::make_cuda_auto_ptr<double>(size_, stream);
        hipMemcpyAsync(qiInv(), cpu_rns_base.inv(), size_ * sizeof(double),
                        hipMemcpyHostToDevice, stream);
    }

    __global__ void decompose_array_uint64(uint64_t *dst, const hipDoubleComplex *src, const DModulus *modulus,
                                           const uint32_t sparse_poly_degree, const uint32_t sparse_ratio,
                                           const uint32_t coeff_mod_size) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < sparse_poly_degree * coeff_mod_size;
             tid += blockDim.x * gridDim.x) {
            size_t twr = tid / sparse_poly_degree;
            size_t coeff_id = tid % sparse_poly_degree;
            DModulus mod = modulus[twr];

            double coeffd;
            if (coeff_id < sparse_poly_degree >> 1) {
                coeffd = round(hipCreal(src[coeff_id]));
            } else {
                coeffd = round(hipCimag(src[coeff_id - (sparse_poly_degree >> 1)]));
            }
            bool is_negative = static_cast<bool>(signbit(coeffd));
            auto coeffu = static_cast<uint64_t>(fabs(coeffd));
            uint32_t index = tid * sparse_ratio;

            uint64_t temp = barrett_reduce_uint64_uint64(coeffu, mod.value(), mod.const_ratio()[1]);

            if (is_negative) {
                temp = mod.value() - temp;
            }

            dst[index] = temp;

            for (uint32_t i = 1; i < sparse_ratio; i++) {
                dst[index + i] = 0;
            }
        }
    }

    __global__ void decompose_array_uint128(uint64_t *dst, const hipDoubleComplex *src, const DModulus *modulus,
                                            const uint32_t sparse_poly_degree, const uint32_t sparse_ratio,
                                            const uint32_t coeff_mod_size) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < sparse_poly_degree * coeff_mod_size;
             tid += blockDim.x * gridDim.x) {
            size_t twr = tid / sparse_poly_degree;
            size_t coeff_id = tid % sparse_poly_degree;
            DModulus mod = modulus[twr];

            double coeffd;
            if (coeff_id < sparse_poly_degree >> 1) {
                coeffd = round(hipCreal(src[coeff_id]));
            } else {
                coeffd = round(hipCimag(src[coeff_id - (sparse_poly_degree >> 1)]));
            }
            bool is_negative = static_cast<bool>(signbit(coeffd));
            coeffd = fabs(coeffd);
            uint64_t coeffu[2] = {
                    static_cast<uint64_t>(fmod(coeffd, two_pow_64_dev)),
                    static_cast<uint64_t>(coeffd / two_pow_64_dev)
            };
            uint32_t index = tid * sparse_ratio;

            uint64_t temp = barrett_reduce_uint128_uint64({coeffu[1], coeffu[0]}, mod.value(), mod.const_ratio());

            if (is_negative) {
                temp = mod.value() - temp;
            }

            dst[index] = temp;

            for (uint32_t i = 1; i < sparse_ratio; i++) {
                dst[index + i] = 0;
            }
        }
    }

    __global__ void decompose_array_uint_slow_first_part(uint64_t *dst, const hipDoubleComplex *src,
                                                         const uint32_t sparse_poly_degree,
                                                         const uint32_t coeff_mod_size) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
             tid < sparse_poly_degree; tid += blockDim.x * gridDim.x) {
            double coeffd;
            if (tid < sparse_poly_degree >> 1)
                coeffd = round(hipCreal(src[tid]));
            else
                coeffd = round(hipCimag(src[tid - (sparse_poly_degree >> 1)]));

            size_t coeff_id = tid * (coeff_mod_size + 1);
            dst[coeff_id + coeff_mod_size] = static_cast<bool>(signbit(coeffd));
            coeffd = fabs(coeffd);
            for (uint32_t i = 0; i < coeff_mod_size; i++) {
                if (coeffd >= 1) {
                    dst[coeff_id + i] = static_cast<uint64_t>(fmod(coeffd, two_pow_64_dev));
                    coeffd /= two_pow_64_dev;
                } else {
                    dst[coeff_id + i] = 0;
                }
            }
        }
    }

    __global__ void decompose_array_uint_slow_second_part(uint64_t *dst, const uint64_t *src, const DModulus *modulus,
                                                          const uint32_t sparse_poly_degree,
                                                          const uint32_t sparse_ratio,
                                                          const uint32_t coeff_mod_size) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < sparse_poly_degree * coeff_mod_size;
             tid += blockDim.x * gridDim.x) {
            size_t twr = tid / sparse_poly_degree;
            size_t coeff_id = (tid % sparse_poly_degree) * (coeff_mod_size + 1);
            DModulus mod = modulus[twr];

            uint128_t temp = {src[coeff_id + coeff_mod_size - 1], 0};
            for (uint32_t i = coeff_mod_size - 1; i--;) {
                temp.lo = src[coeff_id + i];
                temp.hi = barrett_reduce_uint128_uint64(temp, mod.value(), mod.const_ratio());
            }
            // temp.hi holds the final reduction value

            // Save the result modulo i-th prime
            uint32_t index = tid * sparse_ratio;
            if (src[coeff_id + coeff_mod_size]) {
                temp.hi = mod.value() - temp.hi;
            }

            dst[index] = temp.hi;

            for (uint32_t i = 1; i < sparse_ratio; i++) {
                dst[index + i] = 0;
            }
        }
    }

    void DRNSBase::decompose_array(uint64_t *dst, const hipDoubleComplex *src, const uint32_t sparse_poly_degree,
                                   const uint32_t sparse_ratio, const uint32_t max_coeff_bit_count,
                                   const hipStream_t &stream) const {
        uint64_t gridDimGlb = sparse_poly_degree * size() / blockDimGlb.x;
        if (max_coeff_bit_count <= 64) {
            decompose_array_uint64<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                    dst, src, base(), sparse_poly_degree, sparse_ratio,
                    size());
        } else if (max_coeff_bit_count <= 128) {
            decompose_array_uint128<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                    dst, src, base(), sparse_poly_degree, sparse_ratio,
                    size());
        } else {
            auto coeffu = make_cuda_auto_ptr<uint64_t>(sparse_poly_degree * (size() + 1), stream);
            decompose_array_uint_slow_first_part<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                    coeffu.get(), src, sparse_poly_degree, size());
            decompose_array_uint_slow_second_part<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                    dst, coeffu.get(), base(), sparse_poly_degree, sparse_ratio, size());
        }
    }

    __global__ void compose_kernel(hipDoubleComplex *dst, uint64_t *temp_prod_array, uint64_t *acc_mod_array,
                                   const uint64_t *src, const uint32_t size, const DModulus *base_q,
                                   const uint64_t *base_prod, const uint64_t *punctured_prod_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array_shoup,
                                   const uint64_t *upper_half_threshold, const double inv_scale,
                                   const uint32_t coeff_count,
                                   const uint32_t sparse_coeff_count, const uint32_t sparse_ratio) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
             tid < sparse_coeff_count; tid += blockDim.x * gridDim.x) {
            if (size > 1) {
                uint64_t prod;

                for (uint32_t i = 0; i < size; i++) {
                    // [a[j] * hat(q)_j^(-1)]_(q_j)
                    prod = multiply_and_reduce_shoup(src[tid * sparse_ratio + i * coeff_count],
                                                     inv_punctured_prod_mod_base_array[i],
                                                     inv_punctured_prod_mod_base_array_shoup[i], base_q[i].value());

                    // * hat(q)_j over ZZ
                    multiply_uint_uint64(punctured_prod_array + i * size, size, // operand1 and size
                                         prod, // operand2 with uint64_t
                                         temp_prod_array + tid * size); // result and size

                    // accumulation and mod Q over ZZ
                    add_uint_uint_mod(temp_prod_array + tid * size, acc_mod_array + tid * size, base_prod, size,
                                      acc_mod_array + tid * size);
                }
            } else {
                acc_mod_array[tid] = src[tid * sparse_ratio];
            }

            // Create floating-point representations of the multi-precision integer coefficients
            // Scaling instead incorporated above; this can help in cases
            // where otherwise pow(two_pow_64, j) would overflow due to very
            // large coeff_modulus_size and very large scale
            // res[i] = res_accum * inv_scale;
            double res = 0.0;
            double scaled_two_pow_64 = inv_scale;
            uint64_t diff;

            if (is_greater_than_or_equal_uint(acc_mod_array + tid * size, upper_half_threshold, size)) {
                for (uint32_t i = 0; i < size; i++, scaled_two_pow_64 *= two_pow_64_dev) {
                    if (acc_mod_array[tid * size + i] > base_prod[i]) {
                        diff = acc_mod_array[tid * size + i] - base_prod[i];
                        res += diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                    } else {
                        diff = base_prod[i] - acc_mod_array[tid * size + i];
                        res -= diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                    }
                }
            } else {
                for (size_t i = 0; i < size; i++, scaled_two_pow_64 *= two_pow_64_dev) {
                    diff = acc_mod_array[tid * size + i];
                    res += diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                }
            }

            if (tid < sparse_coeff_count >> 1)
                dst[tid].x = res;
            else
                dst[tid - (sparse_coeff_count >> 1)].y = res;
            // TODO: FIXME: Temporary hack to fix bug on AGX Xavier (use printf to block threads), may affect performance
            printf("");
        }
    }

    __global__ void compose_kernel_step1(hipDoubleComplex *dst, uint64_t *temp_prod_array, uint64_t *acc_mod_array,
                                   const uint64_t *src, const uint32_t size, const DModulus *base_q,
                                   const uint64_t *base_prod, const uint64_t *punctured_prod_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array_shoup,
                                   const uint64_t *upper_half_threshold, const double inv_scale,
                                   const uint32_t coeff_count,
                                   const uint32_t sparse_coeff_count, const uint32_t sparse_ratio) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
             tid < sparse_coeff_count; tid += blockDim.x * gridDim.x) {
            if (size > 1) {
                uint64_t prod;

                for (uint32_t i = 0; i < size; i++) {
                    // [a[j] * hat(q)_j^(-1)]_(q_j)
                    prod = multiply_and_reduce_shoup(src[tid * sparse_ratio + i * coeff_count],
                                                     inv_punctured_prod_mod_base_array[i],
                                                     inv_punctured_prod_mod_base_array_shoup[i], base_q[i].value());

                    // * hat(q)_j over ZZ
                    multiply_uint_uint64(punctured_prod_array + i * size, size, // operand1 and size
                                         prod, // operand2 with uint64_t
                                         temp_prod_array + tid * size); // result and size

                    // accumulation and mod Q over ZZ
                    add_uint_uint_mod(temp_prod_array + tid * size, acc_mod_array + tid * size, base_prod, size,
                                      acc_mod_array + tid * size);
                }
            } else {
                acc_mod_array[tid] = src[tid * sparse_ratio];
            }
        }
    }

    __global__ void compose_kernel_step2(hipDoubleComplex *dst, uint64_t *temp_prod_array, uint64_t *acc_mod_array,
                                   const uint64_t *src, const uint32_t size, const DModulus *base_q,
                                   const uint64_t *base_prod, const uint64_t *punctured_prod_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array,
                                   const uint64_t *inv_punctured_prod_mod_base_array_shoup,
                                   const uint64_t *upper_half_threshold, const double inv_scale,
                                   const uint32_t coeff_count,
                                   const uint32_t sparse_coeff_count, const uint32_t sparse_ratio) {
        for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
             tid < sparse_coeff_count; tid += blockDim.x * gridDim.x) {
            // Create floating-point representations of the multi-precision integer coefficients
            // Scaling instead incorporated above; this can help in cases
            // where otherwise pow(two_pow_64, j) would overflow due to very
            // large coeff_modulus_size and very large scale
            // res[i] = res_accum * inv_scale;
            double res = 0.0;
            double scaled_two_pow_64 = inv_scale;
            uint64_t diff;

            if (is_greater_than_or_equal_uint(acc_mod_array + tid * size, upper_half_threshold, size)) {
                for (uint32_t i = 0; i < size; i++, scaled_two_pow_64 *= two_pow_64_dev) {
                    if (acc_mod_array[tid * size + i] > base_prod[i]) {
                        diff = acc_mod_array[tid * size + i] - base_prod[i];
                        res += diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                    } else {
                        diff = base_prod[i] - acc_mod_array[tid * size + i];
                        res -= diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                    }
                }
            } else {
                for (size_t i = 0; i < size; i++, scaled_two_pow_64 *= two_pow_64_dev) {
                    diff = acc_mod_array[tid * size + i];
                    res += diff ? static_cast<double>(diff) * scaled_two_pow_64 : 0.0;
                }
            }

            if (tid < sparse_coeff_count >> 1)
                dst[tid].x = res;
            else
                dst[tid - (sparse_coeff_count >> 1)].y = res;
        }
    }

    __global__ void compose_kernel_step1_1(const uint32_t sparse_ratio, std::size_t coeff_count, std::size_t coeff_modulus_size, std::uint64_t* acc_mod_array) {
        std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < coeff_count && ((idx - 1) & (sparse_ratio - 1)) != sparse_ratio - 1) {
            for (std::size_t j = 0; j < coeff_modulus_size; j++) {
                acc_mod_array[idx * coeff_modulus_size + j] = 0;
            }
        }
    }

    void DRNSBase::compose_array(hipDoubleComplex *dst, const uint64_t *src, const uint64_t *upper_half_threshold,
                                 const double inv_scale, const uint32_t coeff_count, const uint32_t sparse_coeff_count,
                                 const uint32_t sparse_ratio, const hipStream_t &stream) const {
        if (!src) {
            throw invalid_argument("input array cannot be null");
        }

        uint32_t rns_poly_uint64_count = sparse_coeff_count * size();
        auto temp_prod_array = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
        auto acc_mod_array = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
        hipMemsetAsync(acc_mod_array.get(), 0, rns_poly_uint64_count * sizeof(uint64_t), stream);

        uint64_t gridDimGlb = ceil(sparse_coeff_count / blockDimGlb.x);

        compose_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                dst, temp_prod_array.get(), acc_mod_array.get(), src, size(), base(),
                big_modulus(), big_qiHat(), QHatInvModq(), QHatInvModq_shoup(),
                upper_half_threshold, inv_scale, coeff_count, sparse_coeff_count, sparse_ratio);
    }

    void DRNSBase::compose_array(hipDoubleComplex *dst, const uint64_t *src, const uint64_t *upper_half_threshold,
                                 const double inv_scale, const uint32_t coeff_count, const uint32_t sparse_coeff_count,
                                 const uint32_t sparse_ratio, const uint32_t decoding_sparse_ratio, const hipStream_t &stream) const {
        if (!src) {
            throw invalid_argument("input array cannot be null");
        }

        uint32_t rns_poly_uint64_count = sparse_coeff_count * size();
        auto temp_prod_array = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
        auto acc_mod_array = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
        hipMemsetAsync(acc_mod_array.get(), 0, rns_poly_uint64_count * sizeof(uint64_t), stream);

        uint64_t gridDimGlb = ceil(sparse_coeff_count / blockDimGlb.x);

        compose_kernel_step1<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                dst, temp_prod_array.get(), acc_mod_array.get(), src, size(), base(),
                big_modulus(), big_qiHat(), QHatInvModq(), QHatInvModq_shoup(),
                upper_half_threshold, inv_scale, coeff_count, sparse_coeff_count, sparse_ratio);
        
        // Newly added to handle decoding_sparse_slots_ != slots_
        if (decoding_sparse_ratio != 1) {
            int numBlocks = (coeff_count + blockDimGlb.x - 1) / blockDimGlb.x;
            compose_kernel_step1_1<<<numBlocks, blockDimGlb, 0, stream>>>(decoding_sparse_ratio, coeff_count, size(), acc_mod_array.get());
        }
        
        compose_kernel_step2<<<gridDimGlb, blockDimGlb, 0, stream>>>(
                dst, temp_prod_array.get(), acc_mod_array.get(), src, size(), base(),
                big_modulus(), big_qiHat(), QHatInvModq(), QHatInvModq_shoup(),
                upper_half_threshold, inv_scale, coeff_count, sparse_coeff_count, sparse_ratio);
    }
}
